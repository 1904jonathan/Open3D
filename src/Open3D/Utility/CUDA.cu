
#include "CUDA.cuh"

#include <iostream>
using namespace std;

// ----------------------------------------------------------------------------
// Diplay info about the specified device.
// ----------------------------------------------------------------------------
void DeviceInfo(const int& devID)
{
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, devID);

    cout << "GPU Device " << devID << ": ";
    cout << deviceProp.name << ", ";
    cout << "CUDA ";
    cout << deviceProp.major << ".";
    cout << deviceProp.minor << endl;
    cout << endl;
}

// ---------------------------------------------------------------------------
// Alocate host memory and perform validation.
// ---------------------------------------------------------------------------
bool AlocateHstMemory(float** h, const int& numElements, const string& name)
{
    size_t size = numElements * sizeof(float);

    *h = (float *)malloc(size);

    if (*h != NULL)
        return true;

    cout << "Failed to allocate host memory: " << name << endl;

    return false;
}

// ---------------------------------------------------------------------------
// Alocate device memory and perform validation.
// ---------------------------------------------------------------------------
bool AlocateDevMemory(float** d, const int& numElements, const string& name)
{
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(float);

    status = hipMalloc((void **)d, size);

    if (status == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to allocate device memory: " << name << endl;

    return false;
}

// ---------------------------------------------------------------------------
// Initialize host memory.
// ---------------------------------------------------------------------------
void RandInit(float* h, const int& numElements)
{
    for (int i = 0; i < numElements; ++i)
        h[i] = rand()/(float)RAND_MAX;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool CopyHst2DevMemory(float* h, float* d, const int& numElements)
{
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(float);

    cout << "Copy host memory to the CUDA device." << endl;
    status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    if (status == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to copy host memory to the CUDA device." << endl;

    return false;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool CopyDev2HstMemory(float* d, float* h, const int& numElements)
{
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(float);

    cout << "Copy device memory to the host." << endl;
    status = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    if (status == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to copy device memory to the host." << endl;

    return false;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool freeDev(float** d, const string& name)
{
    hipError_t status = hipSuccess;

    status = hipFree(*d);

    if (status == hipSuccess)
    {
        *d = NULL;
        return true;
    }

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to free device vector" << name << endl;

    return false;
}
