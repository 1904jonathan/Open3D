
#include "CUDA.cuh"

#include <iostream>
using namespace std;

// ----------------------------------------------------------------------------
// Diplay info about the specified device.
// ----------------------------------------------------------------------------
void DeviceInfo(const int& devID)
{
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, devID);

    cout << "GPU Device " << devID << ": ";
    cout << deviceProp.name << ", ";
    cout << "CUDA ";
    cout << deviceProp.major << ".";
    cout << deviceProp.minor << endl;
    cout << endl;
}

// ---------------------------------------------------------------------------
// Alocate host memory and perform validation.
// ---------------------------------------------------------------------------
bool AlocateHstMemory(double** h, const int& numElements, const string& name)
{
    size_t size = numElements * sizeof(double);

    *h = (double *)malloc(size);

    if (*h != NULL)
        return true;

    cout << "Failed to allocate host memory: " << name << endl;

    return false;
}

// ---------------------------------------------------------------------------
// Alocate device memory and perform validation.
// ---------------------------------------------------------------------------
bool AlocateDevMemory(double** d, const int& numElements, const string& name)
{
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMalloc((void **)d, size);

    if (status == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to allocate device memory: " << name << endl;

    return false;
}

// ---------------------------------------------------------------------------
// Initialize host memory.
// ---------------------------------------------------------------------------
void RandInit(double* h, const int& numElements)
{
    for (int i = 0; i < numElements; ++i)
        h[i] = rand()/(double)RAND_MAX;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool CopyHst2DevMemory(double* h, double* d, const int& numElements)
{
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    if (status == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to copy host memory to the CUDA device." << endl;

    return false;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool CopyDev2HstMemory(double* d, double* h, const int& numElements)
{
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    if (status == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to copy device memory to the host." << endl;

    return false;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool freeDev(double** d, const string& name)
{
    hipError_t status = hipSuccess;

    status = hipFree(*d);

    if (status == hipSuccess)
    {
        *d = NULL;
        return true;
    }

    cout << "status: " << hipGetErrorString(status) << endl;
    cout << "Failed to free device vector" << name << endl;

    return false;
}
