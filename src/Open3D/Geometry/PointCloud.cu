#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <stdio.h>

#include "Open3D/Types/Vector3f.h"
#include "Open3D/Types/Matrix3f.h"
using namespace open3d;

#include "Open3D/Utility/CUDA.cuh"

#include <iostream>
using namespace std;

// ---------------------------------------------------------------------------
// dummy kernel
// ---------------------------------------------------------------------------
__global__ void dummy(float* data, int size, float* output) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    Vector3f* points = (Vector3f*)data;
    Matrix3f* cumulants = (Matrix3f*)output;

    Vector3f p = points[gid];
    Matrix3f c = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

    c[0][0] += p[0];
    c[0][1] += p[1];
    c[0][2] += p[2];
    c[1][0] += p[0] * p[0];
    c[1][1] += p[0] * p[1];
    c[1][2] += p[0] * p[2];
    c[2][0] += p[1] * p[1];
    c[2][1] += p[1] * p[2];
    c[2][2] += p[2] * p[2];

    printf("%4d: %+6.3f %+6.3f %+6.3f\n      %+6.3f %+6.3f %+6.3f\n      %+6.3f %+6.3f %+6.3f\n",
        gid, c[0][0], c[0][1], c[0][2], c[1][0], c[1][1], c[1][2], c[2][0], c[2][1], c[2][2]);

    cumulants[gid][0][0] = c[0][0];
    cumulants[gid][0][1] = c[0][1];
    cumulants[gid][0][2] = c[0][2];

    cumulants[gid][1][0] = c[1][0];
    cumulants[gid][1][1] = c[1][1];
    cumulants[gid][1][2] = c[1][2];

    cumulants[gid][2][0] = c[2][0];
    cumulants[gid][2][1] = c[2][1];
    cumulants[gid][2][2] = c[2][2];
}

// ---------------------------------------------------------------------------
// helper function calls the dummy kernel
// ---------------------------------------------------------------------------
void dummyHost() {
    cout << "dummyHost::START" << endl;
    cout << endl;

    // Error code to check return values for CUDA calls
    hipError_t status = hipSuccess;

    // nr. of dimensions
    int nrPoints = 1 << 8;
    cout << "nr. of points:" << nrPoints << endl;

    int inputSize = nrPoints * Vector3f::SIZE;
    int outputSize = nrPoints * Matrix3f::SIZE;

    // host memory
    float *h_A = NULL;
    float *h_C = NULL;

    // device memory
    float *d_A = NULL;
    float *d_C = NULL;

    if (!AlocateHstMemory(&h_A, inputSize, "h_A")) exit(1);
    if (!AlocateHstMemory(&h_C, outputSize, "h_C")) exit(1);

    RandInit(h_A, inputSize);

    if (!AlocateDevMemory(&d_A, inputSize, "d_A")) exit(1);
    if (!AlocateDevMemory(&d_C, outputSize, "d_C")) exit(1);

    // Copy input to the device
    CopyHst2DevMemory(h_A, d_A, inputSize);

    // Launch the dummy CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(nrPoints + threadsPerBlock - 1) / threadsPerBlock;

    cout << "CUDA kernel launch with " << blocksPerGrid;
    cout << " blocks of " << threadsPerBlock << " threads" << endl;

    dummy<<<blocksPerGrid, threadsPerBlock>>>(d_A, nrPoints, d_C);
    hipDeviceSynchronize();
    status = hipGetLastError();

    if (status != hipSuccess)
    {
        cout << "status: " << hipGetErrorString(status) << endl;
        cout << "Failed to launch vectorAdd kernel" << endl;
        exit(1);
    }

    // Copy results to the host
    CopyDev2HstMemory(d_C, h_C, outputSize);

    Matrix3f* cumulants = (Matrix3f*)h_C;
    cout << endl;
    cout << endl;
    for (int i = 0; i < nrPoints; i++)
    {
        Matrix3f c = cumulants[i];
        printf("%4d: %+6.3f %+6.3f %+6.3f\n      %+6.3f %+6.3f %+6.3f\n      %+6.3f %+6.3f %+6.3f\n",
        i, c[0][0], c[0][1], c[0][2], c[1][0], c[1][1], c[1][2], c[2][0], c[2][1], c[2][2]);
    }

    // Free device global memory
    freeDev(&d_A, "d_A");
    freeDev(&d_C, "d_C");

    // Free host memory
    free(h_A);
    free(h_C);

    cout << endl;
    cout << "dummyHost::END" << endl;
}
