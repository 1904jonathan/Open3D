#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include "Open3D/Types/Vector3f.h"
using namespace open3d;

__global__ void dummy(Vector3f* points, int size) {
    // global thread id x
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // nr points / thread
    int chunkSize = size / gridDim.x / blockDim.x;

    // index 0 of points corresponding to current thread
    int globalOffset = gid * chunkSize;

    Vector3f* localPoints = &points[globalOffset];

    Vector3f point = localPoints[gid];

    point[0] + point[1] + point[2];
}

void dummy() {
}
