#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <stdio.h>

#include "Open3D/Types/Vector3f.h"
#include "Open3D/Types/Matrix3f.h"
using namespace open3d;

#include <iostream>
using namespace std;

// ---------------------------------------------------------------------------
// dummy kernel
// ---------------------------------------------------------------------------
__global__ void dummy(float* data, int nrPoints, float* output) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    Vector3f* points = (Vector3f*)data;
    Matrix3f* cumulants = (Matrix3f*)output;

    Vector3f p = points[gid];
    Matrix3f c = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

    c[0][0] += p[0];
    c[0][1] += p[1];
    c[0][2] += p[2];
    c[1][0] += p[0] * p[0];
    c[1][1] += p[0] * p[1];
    c[1][2] += p[0] * p[2];
    c[2][0] += p[1] * p[1];
    c[2][1] += p[1] * p[2];
    c[2][2] += p[2] * p[2];

    printf("%4d: %+6.3f %+6.3f %+6.3f\n      %+6.3f %+6.3f %+6.3f\n      %+6.3f %+6.3f %+6.3f\n",
        gid, c[0][0], c[0][1], c[0][2], c[1][0], c[1][1], c[1][2], c[2][0], c[2][1], c[2][2]);

    cumulants[gid][0][0] = c[0][0];
    cumulants[gid][0][1] = c[0][1];
    cumulants[gid][0][2] = c[0][2];

    cumulants[gid][1][0] = c[1][0];
    cumulants[gid][1][1] = c[1][1];
    cumulants[gid][1][2] = c[1][2];

    cumulants[gid][2][0] = c[2][0];
    cumulants[gid][2][1] = c[2][1];
    cumulants[gid][2][2] = c[2][2];
}

// ---------------------------------------------------------------------------
// helper function calls the dummy kernel
// ---------------------------------------------------------------------------
void dummyGPU(float* const d_A, const int& nrPoints, float* const d_C) {
    cout << "dummyGPU::START" << endl;
    cout << endl;

    // Launch the dummy CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(nrPoints + threadsPerBlock - 1) / threadsPerBlock;

    cout << "CUDA kernel launch with " << blocksPerGrid;
    cout << " blocks of " << threadsPerBlock << " threads" << endl;

    dummy<<<blocksPerGrid, threadsPerBlock>>>(d_A, nrPoints, d_C);
    hipDeviceSynchronize();

    cout << endl;
    cout << "dummyGPU::END" << endl;
}
