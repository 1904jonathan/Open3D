#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2020 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------
//

#define EIGEN_USE_GPU
#include "FixedRadiusSearchOpKernel.h"
#include "Open3D/Core/CUDAUtils.h"
#include "Open3D/ML/Misc/Detail/FixedRadiusSearch.cuh"

using namespace open3d;
using namespace open3d::ml::detail;
using namespace fixed_radius_search_opkernel;
using namespace tensorflow;

template <class T>
class FixedRadiusSearchOpKernelCUDA : public FixedRadiusSearchOpKernel {
public:
    explicit FixedRadiusSearchOpKernelCUDA(OpKernelConstruction* construction)
        : FixedRadiusSearchOpKernel(construction) {
        texture_alignment = GetCUDACurrentDeviceTextureAlignment();
    }

    void Kernel(tensorflow::OpKernelContext* context,
                const tensorflow::Tensor& points,
                const tensorflow::Tensor& queries,
                const tensorflow::Tensor& radius,
                const size_t hash_table_size,
                tensorflow::Tensor& query_neighbors_row_splits) {
        auto device = context->eigen_gpu_device();

        OutputAllocator<T> output_allocator(context);

        Tensor hash_table_row_splits;
        TensorShape hash_table_row_splits_shape({ssize_t(hash_table_size + 1)});
        OP_REQUIRES_OK(context,
                       context->allocate_temp(DataTypeToEnum<uint32_t>::v(),
                                              hash_table_row_splits_shape,
                                              &hash_table_row_splits));

        Tensor hash_table_index;
        TensorShape hash_table_index_shape({points.shape().dim_size(0)});
        OP_REQUIRES_OK(context,
                       context->allocate_temp(DataTypeToEnum<uint32_t>::v(),
                                              hash_table_index_shape,
                                              &hash_table_index));

        void* temp_ptr = nullptr;
        size_t temp_size = 0;
        size_t temp_size_hash_table = 0;

        // determine temp_size
        BuildSpatialHashTableCUDA(device.stream(), temp_ptr,
                                  temp_size_hash_table, texture_alignment,
                                  points.shape().dim_size(0),
                                  points.flat<T>().data(), radius.scalar<T>()(),
                                  hash_table_size + 1,
                                  hash_table_row_splits.flat<uint32_t>().data(),
                                  hash_table_index.flat<uint32_t>().data());

        FixedRadiusSearchCUDA(
                device.stream(), temp_ptr, temp_size, texture_alignment,
                (int64_t*)query_neighbors_row_splits.flat<int64>().data(),
                points.shape().dim_size(0), points.flat<T>().data(),
                queries.shape().dim_size(0), queries.flat<T>().data(),
                radius.scalar<T>()(), hash_table_row_splits.shape().dim_size(0),
                hash_table_row_splits.flat<uint32_t>().data(),
                hash_table_index.flat<uint32_t>().data(), metric,
                ignore_query_point, return_distances, output_allocator);

        temp_size = std::max(temp_size, temp_size_hash_table);
        Tensor temp_tensor;
        TensorShape temp_shape({ssize_t(temp_size)});
        OP_REQUIRES_OK(context,
                       context->allocate_temp(DataTypeToEnum<uint8_t>::v(),
                                              temp_shape, &temp_tensor));
        temp_ptr = temp_tensor.flat<uint8_t>().data();

        // actually run the search
        BuildSpatialHashTableCUDA(device.stream(), temp_ptr,
                                  temp_size_hash_table, texture_alignment,
                                  points.shape().dim_size(0),
                                  points.flat<T>().data(), radius.scalar<T>()(),
                                  hash_table_size + 1,
                                  hash_table_row_splits.flat<uint32_t>().data(),
                                  hash_table_index.flat<uint32_t>().data());

        FixedRadiusSearchCUDA(
                device.stream(), temp_ptr, temp_size, texture_alignment,
                (int64_t*)query_neighbors_row_splits.flat<int64>().data(),
                points.shape().dim_size(0), points.flat<T>().data(),
                queries.shape().dim_size(0), queries.flat<T>().data(),
                radius.scalar<T>()(), hash_table_row_splits.shape().dim_size(0),
                hash_table_row_splits.flat<uint32_t>().data(),
                hash_table_index.flat<uint32_t>().data(), metric,
                ignore_query_point, return_distances, output_allocator);
    }

private:
    int texture_alignment;
};

#define REG_KB(type)                                                       \
    REGISTER_KERNEL_BUILDER(Name("Open3DFixedRadiusSearch")                \
                                    .Device(DEVICE_GPU)                    \
                                    .TypeConstraint<type>("T")             \
                                    .HostMemory("radius")                  \
                                    .HostMemory("hash_table_size_factor"), \
                            FixedRadiusSearchOpKernelCUDA<type>);
REG_KB(float)
#undef REG_KB
