// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "open3d/core/Tensor.h"
#include "open3d/core/kernel/CUDALauncher.cuh"
#include "open3d/t/pipelines/kernel/RegistrationImpl.h"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {
namespace registration {

inline void ReduceAndSolve6x6(float *A_reduction,
                              int64_t n,
                              core::Tensor &delta,
                              double &residual,
                              int64_t &count,
                              const core::Device &device) {
    core::Tensor output_29 =
            core::Tensor::Empty({29}, core::Dtype::Float32, device);
    float *output_29_data = output_29.GetDataPtr<float>();

    // Reduction of {29, N} to {29}.
    for (int i = 0; i < 29; i++) {
        // Determine temporary device storage requirements.
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               A_reduction + i * n, output_29_data + i, n);
        // Allocate temporary storage.
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run sum-reduction.
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               A_reduction + i * n, output_29_data + i, n);
        hipFree(d_temp_storage);
    }

    DecodeAndSolve6x6(output_29, delta, residual, count);
}

void ComputePosePointToPlaneCUDA(
        const core::Tensor &source_points,
        const core::Tensor &target_points,
        const core::Tensor &target_normals,
        const std::pair<core::Tensor, core::Tensor> &corres,
        core::Tensor &pose,
        double &residual,
        int64_t &count,
        const core::Dtype &dtype,
        const core::Device &device) {
    const float *source_points_ptr = source_points.GetDataPtr<float>();
    const float *target_points_ptr = target_points.GetDataPtr<float>();
    const float *target_normals_ptr = target_normals.GetDataPtr<float>();
    const int64_t *correspondences_first = corres.first.GetDataPtr<int64_t>();
    const float *correspondences_second = corres.second.GetDataPtr<float>();

    int n = corres.first.GetLength();

    // A_29xN is a {29, N} shaped tensor, which is later reduced to {29} where
    // [0, 20] elements are used to construct {6,6} shaped symmetric AtA matrix,
    // and [21, 26] elements are used to construct {6} AtB matrix, [27] is
    // residual or squared_error, [28] is number of correspondences or count.
    core::Tensor A_29xN =
            core::Tensor::Empty({29, n}, core::Dtype::Float32, device);
    float *A_reduction = A_29xN.GetDataPtr<float>();

    core::kernel::CUDALauncher::LaunchGeneralKernel(
            n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float J_ij[6];
                float r;

                bool valid = GetJacobianPointToPlane<float>(
                        workload_idx, source_points_ptr, target_points_ptr,
                        target_normals_ptr, correspondences_first, J_ij, r);

                if (valid) {
                    A_reduction[workload_idx] = J_ij[0] * J_ij[0];
                    A_reduction[n + workload_idx] = J_ij[1] * J_ij[0];
                    A_reduction[n * 2 + workload_idx] = J_ij[1] * J_ij[1];
                    A_reduction[n * 3 + workload_idx] = J_ij[2] * J_ij[0];
                    A_reduction[n * 4 + workload_idx] = J_ij[2] * J_ij[1];
                    A_reduction[n * 5 + workload_idx] = J_ij[2] * J_ij[2];
                    A_reduction[n * 6 + workload_idx] = J_ij[3] * J_ij[0];
                    A_reduction[n * 7 + workload_idx] = J_ij[3] * J_ij[1];
                    A_reduction[n * 8 + workload_idx] = J_ij[3] * J_ij[2];
                    A_reduction[n * 9 + workload_idx] = J_ij[3] * J_ij[3];
                    A_reduction[n * 10 + workload_idx] = J_ij[4] * J_ij[0];
                    A_reduction[n * 11 + workload_idx] = J_ij[4] * J_ij[1];
                    A_reduction[n * 12 + workload_idx] = J_ij[4] * J_ij[2];
                    A_reduction[n * 13 + workload_idx] = J_ij[4] * J_ij[3];
                    A_reduction[n * 14 + workload_idx] = J_ij[4] * J_ij[4];
                    A_reduction[n * 15 + workload_idx] = J_ij[5] * J_ij[0];
                    A_reduction[n * 16 + workload_idx] = J_ij[5] * J_ij[1];
                    A_reduction[n * 17 + workload_idx] = J_ij[5] * J_ij[2];
                    A_reduction[n * 18 + workload_idx] = J_ij[5] * J_ij[3];
                    A_reduction[n * 19 + workload_idx] = J_ij[5] * J_ij[4];
                    A_reduction[n * 20 + workload_idx] = J_ij[5] * J_ij[5];

                    A_reduction[n * 21 + workload_idx] = J_ij[0] * r;
                    A_reduction[n * 22 + workload_idx] = J_ij[1] * r;
                    A_reduction[n * 23 + workload_idx] = J_ij[2] * r;
                    A_reduction[n * 24 + workload_idx] = J_ij[3] * r;
                    A_reduction[n * 25 + workload_idx] = J_ij[4] * r;
                    A_reduction[n * 26 + workload_idx] = J_ij[5] * r;

                    A_reduction[n * 27 + workload_idx] =
                            correspondences_second[workload_idx];
                    A_reduction[n * 28 + workload_idx] = 1;

                } else {
                    A_reduction[n + workload_idx] = 0;
                    A_reduction[n * 2 + workload_idx] = 0;
                    A_reduction[n * 3 + workload_idx] = 0;
                    A_reduction[n * 4 + workload_idx] = 0;
                    A_reduction[n * 5 + workload_idx] = 0;
                    A_reduction[n * 6 + workload_idx] = 0;
                    A_reduction[n * 7 + workload_idx] = 0;
                    A_reduction[n * 8 + workload_idx] = 0;
                    A_reduction[n * 9 + workload_idx] = 0;
                    A_reduction[n * 10 + workload_idx] = 0;
                    A_reduction[n * 11 + workload_idx] = 0;
                    A_reduction[n * 12 + workload_idx] = 0;
                    A_reduction[n * 13 + workload_idx] = 0;
                    A_reduction[n * 14 + workload_idx] = 0;
                    A_reduction[n * 15 + workload_idx] = 0;
                    A_reduction[n * 16 + workload_idx] = 0;
                    A_reduction[n * 17 + workload_idx] = 0;
                    A_reduction[n * 18 + workload_idx] = 0;
                    A_reduction[n * 19 + workload_idx] = 0;
                    A_reduction[n * 20 + workload_idx] = 0;

                    A_reduction[n * 21 + workload_idx] = 0;
                    A_reduction[n * 22 + workload_idx] = 0;
                    A_reduction[n * 23 + workload_idx] = 0;
                    A_reduction[n * 24 + workload_idx] = 0;
                    A_reduction[n * 25 + workload_idx] = 0;
                    A_reduction[n * 26 + workload_idx] = 0;

                    A_reduction[n * 27 + workload_idx] = 0;
                    A_reduction[n * 28 + workload_idx] = 0;
                }
            });

    ReduceAndSolve6x6(A_reduction, n, pose, residual, count, device);
}

}  // namespace registration
}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
