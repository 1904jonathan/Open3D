#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/kernel/CUDALauncher.cuh"
#include "open3d/t/pipelines/kernel/ComputeTransformImpl.h"
#include "open3d/t/pipelines/kernel/Reduction6x6Impl.cuh"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"
#include "open3d/utility/Timer.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {

const int kThread1DUnit = 256;

template <typename scalar_t, class funct_t>
__global__ void ComputePosePointToPlaneCUDAKernel(
        const scalar_t *source_points_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const int64_t *correspondences_second,
        const int n,
        scalar_t *global_sum,
        funct_t op) {
    __shared__ scalar_t local_sum0[kThread1DUnit];
    __shared__ scalar_t local_sum1[kThread1DUnit];
    __shared__ scalar_t local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    scalar_t J[6] = {0}, reduction[29] = {0};
    scalar_t r = 0;

    bool valid = GetJacobianPointToPlane<scalar_t>(
            workload_idx, source_points_ptr, target_points_ptr,
            target_normals_ptr, correspondences_second, J, r);

    scalar_t w = op(r);

    printf(" residual: %lf, weight: %lf", (double)r, (double)w);

    if (valid) {
        // Dump J, r into JtJ and Jtr
        int offset = 0;
        for (int i = 0; i < 6; ++i) {
            for (int j = 0; j <= i; ++j) {
                reduction[offset++] = J[i] * J[j];
            }
        }
        for (int i = 0; i < 6; ++i) {
            reduction[offset++] = J[i] * r;
        }
        reduction[offset++] = r * r;
        reduction[offset++] = valid;
    }

    ReduceSum6x6LinearSystem<scalar_t, kThread1DUnit>(tid, valid, reduction,
                                                   local_sum0, local_sum1,
                                                   local_sum2, global_sum);
}

void ComputePosePointToPlaneCUDA(const core::Tensor &source_points,
                                 const core::Tensor &target_points,
                                 const core::Tensor &target_normals,
                                 const core::Tensor &correspondence_indices,
                                 core::Tensor &pose,
                                 float &residual,
                                 int &inlier_count,
                                 const core::Dtype &dtype,
                                 const core::Device &device) {
    int n = source_points.GetLength();


    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {

        auto robust_kernel = [=] OPEN3D_HOST_DEVICE(scalar_t r) -> scalar_t {
            return 2 * r;
        };

        core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
        scalar_t *global_sum_ptr = global_sum.GetDataPtr<scalar_t>();

        const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
        const dim3 threads(kThread1DUnit);

        ComputePosePointToPlaneCUDAKernel<<<blocks, threads>>>(
                source_points.GetDataPtr<scalar_t>(),
                target_points.GetDataPtr<scalar_t>(),
                target_normals.GetDataPtr<scalar_t>(),
                correspondence_indices.GetDataPtr<int64_t>(), n, global_sum_ptr,
                robust_kernel);

        OPEN3D_CUDA_CHECK(hipDeviceSynchronize());

        DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
    });
}

}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
