#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Dispatch.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/kernel/CUDALauncher.cuh"
#include "open3d/t/geometry/kernel/GeometryIndexer.h"
#include "open3d/t/geometry/kernel/GeometryMacros.h"
#include "open3d/t/pipelines/kernel/RGBDOdometryImpl.h"
#include "open3d/t/pipelines/kernel/RGBDOdometryJacobianImpl.h"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {
namespace odometry {

void ReduceAndSolve6x6(float* A_reduction,
                       core::Tensor& delta,
                       core::Tensor& residual,
                       int64_t n,
                       const core::Device& device) {
    core::Tensor output_29 =
            core::Tensor::Empty({29}, core::Dtype::Float32, device);
    float* output_29_data = output_29.GetDataPtr<float>();

    // Reduction of {29, N} to {29}.
    for (int i = 0; i < 29; i++) {
        // Determine temporary device storage requirements.
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               A_reduction + i * n, output_29_data + i, n);
        // Allocate temporary storage.
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run sum-reduction.
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               A_reduction + i * n, output_29_data + i, n);
        hipFree(d_temp_storage);
    }

    DecodeAndSolve6x6(output_29, delta, residual);
}

template <typename T>
__device__ inline void WarpReduceSum(volatile T* local_sum, const int tid) {
    local_sum[tid] += local_sum[tid + 32];
    local_sum[tid] += local_sum[tid + 16];
    local_sum[tid] += local_sum[tid + 8];
    local_sum[tid] += local_sum[tid + 4];
    local_sum[tid] += local_sum[tid + 2];
    local_sum[tid] += local_sum[tid + 1];
}

template <typename T, size_t BLOCK_SIZE>
__device__ inline void BlockReduceSum(const int tid, volatile T* local_sum) {
    if (BLOCK_SIZE >= 512) {
        if (tid < 256) {
            local_sum[tid] += local_sum[tid + 256];
        }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256) {
        if (tid < 128) {
            local_sum[tid] += local_sum[tid + 128];
        }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128) {
        if (tid < 64) {
            local_sum[tid] += local_sum[tid + 64];
        }
        __syncthreads();
    }
    if (tid < 32) {
        WarpReduceSum<T>(local_sum, tid);
    }
}

template <typename T, size_t BLOCK_SIZE>
__device__ inline void BlockReduceSum(const int tid,
                                      volatile T* local_sum0,
                                      volatile T* local_sum1) {
    if (BLOCK_SIZE >= 512) {
        if (tid < 256) {
            local_sum0[tid] += local_sum0[tid + 256];
            local_sum1[tid] += local_sum1[tid + 256];
        }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256) {
        if (tid < 128) {
            local_sum0[tid] += local_sum0[tid + 128];
            local_sum1[tid] += local_sum1[tid + 128];
        }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128) {
        if (tid < 64) {
            local_sum0[tid] += local_sum0[tid + 64];
            local_sum1[tid] += local_sum1[tid + 64];
        }
        __syncthreads();
    }

    if (tid < 32) {
        WarpReduceSum<float>(local_sum0, tid);
        WarpReduceSum<float>(local_sum1, tid);
    }
}

template <typename T, size_t BLOCK_SIZE>
__device__ inline void BlockReduceSum(const int tid,
                                      volatile T* local_sum0,
                                      volatile T* local_sum1,
                                      volatile T* local_sum2) {
    if (BLOCK_SIZE >= 512) {
        if (tid < 256) {
            local_sum0[tid] += local_sum0[tid + 256];
            local_sum1[tid] += local_sum1[tid + 256];
            local_sum2[tid] += local_sum2[tid + 256];
        }
        __syncthreads();
    }

    if (BLOCK_SIZE >= 256) {
        if (tid < 128) {
            local_sum0[tid] += local_sum0[tid + 128];
            local_sum1[tid] += local_sum1[tid + 128];
            local_sum2[tid] += local_sum2[tid + 128];
        }
        __syncthreads();
    }

    if (BLOCK_SIZE >= 128) {
        if (tid < 64) {
            local_sum0[tid] += local_sum0[tid + 64];
            local_sum1[tid] += local_sum1[tid + 64];
            local_sum2[tid] += local_sum2[tid + 64];
        }
        __syncthreads();
    }

    if (tid < 32) {
        WarpReduceSum<float>(local_sum0, tid);
        WarpReduceSum<float>(local_sum1, tid);
        WarpReduceSum<float>(local_sum2, tid);
    }
}

__global__ void ComputePosePointToPlaneCUDAKernel(
        NDArrayIndexer source_vertex_indexer,
        NDArrayIndexer target_vertex_indexer,
        NDArrayIndexer target_normal_indexer,
        TransformIndexer ti,
        float* global_sum,
        int rows,
        int cols,
        float depth_diff) {
    const int kBlockSize = 256;
    __shared__ float local_sum0[kBlockSize];
    __shared__ float local_sum1[kBlockSize];
    __shared__ float local_sum2[kBlockSize];

    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    if (y >= rows || x >= cols) return;

    float J[6] = {0}, reduction[21 + 6 + 2];
    float r = 0;
    bool valid = GetJacobianPointToPlane(
            x, y, depth_diff, source_vertex_indexer, target_vertex_indexer,
            target_normal_indexer, ti, J, r);

    // Dump J, r into JtJ and Jtr
    int offset = 0;
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j <= i; ++j) {
            reduction[offset++] = J[i] * J[j];
        }
    }
    for (int i = 0; i < 6; ++i) {
        reduction[offset++] = J[i] * r;
    }
    reduction[offset++] = r * r;
    reduction[offset++] = valid;

    // Sum reduction: JtJ(21) and Jtr(6)
    for (size_t i = 0; i < 27; i += 3) {
        local_sum0[tid] = valid ? reduction[i + 0] : 0;
        local_sum1[tid] = valid ? reduction[i + 1] : 0;
        local_sum2[tid] = valid ? reduction[i + 2] : 0;
        __syncthreads();

        BlockReduceSum<float, kBlockSize>(tid, local_sum0, local_sum1,
                                          local_sum2);

        if (tid == 0) {
            atomicAdd(&global_sum[i + 0], local_sum0[0]);
            atomicAdd(&global_sum[i + 1], local_sum1[0]);
            atomicAdd(&global_sum[i + 2], local_sum2[0]);
        }
        __syncthreads();
    }

    // Sum reduction: residual(1) and inlier(1)
    {
        local_sum0[tid] = valid ? reduction[27] : 0;
        local_sum1[tid] = valid ? reduction[28] : 0;
        __syncthreads();

        BlockReduceSum<float, kBlockSize>(tid, local_sum0, local_sum1);
        if (tid == 0) {
            atomicAdd(&global_sum[27], local_sum0[0]);
            atomicAdd(&global_sum[28], local_sum1[0]);
        }
        __syncthreads();
    }
}

void ComputePosePointToPlaneCUDA(const core::Tensor& source_vertex_map,
                                 const core::Tensor& target_vertex_map,
                                 const core::Tensor& target_normal_map,
                                 const core::Tensor& intrinsics,
                                 const core::Tensor& init_source_to_target,
                                 core::Tensor& delta,
                                 core::Tensor& residual,
                                 float depth_diff) {
    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);
    NDArrayIndexer target_vertex_indexer(target_vertex_map, 2);
    NDArrayIndexer target_normal_indexer(target_normal_map, 2);

    core::Device device = source_vertex_map.GetDevice();

    core::Tensor trans = init_source_to_target;
    TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);

    // A_29xN is a {29, N} shaped tensor, which is later reduced to {29} where
    // [0, 20] elements are used to construct {6,6} shaped symmetric AtA
    // matrix, [21, 26] elements are used to construct {6} AtB matrix, element
    // [27] stores residual and element [28] stores count.
    core::Tensor global_sum =
            core::Tensor::Zeros({29}, core::Dtype::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    const int kThreadSize = 16;
    const dim3 blocks((cols + kThreadSize - 1) / kThreadSize,
                      (rows + kThreadSize - 1) / kThreadSize);
    const dim3 threads(kThreadSize, kThreadSize);
    ComputePosePointToPlaneCUDAKernel<<<blocks, threads>>>(
            source_vertex_indexer, target_vertex_indexer, target_normal_indexer,
            ti, global_sum_ptr, rows, cols, depth_diff);
    OPEN3D_CUDA_CHECK(hipDeviceSynchronize());
    DecodeAndSolve6x6(global_sum, delta, residual);
}

void ComputePoseIntensityCUDA(const core::Tensor& source_depth,
                              const core::Tensor& target_depth,
                              const core::Tensor& source_intensity,
                              const core::Tensor& target_intensity,
                              const core::Tensor& target_intensity_dx,
                              const core::Tensor& target_intensity_dy,
                              const core::Tensor& source_vertex_map,
                              const core::Tensor& intrinsics,
                              const core::Tensor& init_source_to_target,
                              core::Tensor& delta,
                              core::Tensor& residual,
                              float depth_diff) {
    NDArrayIndexer source_depth_indexer(source_depth, 2);
    NDArrayIndexer target_depth_indexer(target_depth, 2);

    NDArrayIndexer source_intensity_indexer(source_intensity, 2);
    NDArrayIndexer target_intensity_indexer(target_intensity, 2);

    NDArrayIndexer target_intensity_dx_indexer(target_intensity_dx, 2);
    NDArrayIndexer target_intensity_dy_indexer(target_intensity_dy, 2);

    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);

    core::Device device = source_vertex_map.GetDevice();
    core::Tensor trans = init_source_to_target;
    t::geometry::kernel::TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);
    const int64_t n = rows * cols;

    // A_29xN is a {29, N} shaped tensor, which is later reduced to
    // {29} where [0, 20] elements are used to construct {6,6} shaped
    // symmetric AtA matrix, [21, 26] elements are used to construct {6} AtB
    // matrix, element [27] stores residual and element [28] stores count.
    core::Tensor A_29xN =
            core::Tensor::Empty({29, n}, core::Dtype::Float32, device);
    float* A_reduction = A_29xN.GetDataPtr<float>();

    core::kernel::CUDALauncher::LaunchGeneralKernel(
            n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float J_I[6];
                float r_I;

                bool valid = GetJacobianIntensity(
                        workload_idx, cols, depth_diff, source_depth_indexer,
                        target_depth_indexer, source_intensity_indexer,
                        target_intensity_indexer, target_intensity_dx_indexer,
                        target_intensity_dy_indexer, source_vertex_indexer, ti,
                        J_I, r_I);

                if (valid) {
                    for (int i = 0, j = 0; j < 6; j++) {
                        for (int k = 0; k <= j; k++) {
                            A_reduction[n * i + workload_idx] = J_I[j] * J_I[k];
                            i++;
                        }
                        A_reduction[n * (21 + j) + workload_idx] = J_I[j] * r_I;
                    }
                    A_reduction[n * 27 + workload_idx] = r_I * r_I;
                    A_reduction[n * 28 + workload_idx] = 1;
                } else {
                    for (int i = 0; i < 29; i++) {
                        A_reduction[n * i + workload_idx] = 0;
                    }
                }
            });

    ReduceAndSolve6x6(A_reduction, delta, residual, n, device);
}

void ComputePoseHybridCUDA(const core::Tensor& source_depth,
                           const core::Tensor& target_depth,
                           const core::Tensor& source_intensity,
                           const core::Tensor& target_intensity,
                           const core::Tensor& target_depth_dx,
                           const core::Tensor& target_depth_dy,
                           const core::Tensor& target_intensity_dx,
                           const core::Tensor& target_intensity_dy,
                           const core::Tensor& source_vertex_map,
                           const core::Tensor& intrinsics,
                           const core::Tensor& init_source_to_target,
                           core::Tensor& delta,
                           core::Tensor& residual,
                           float depth_diff) {
    NDArrayIndexer source_depth_indexer(source_depth, 2);
    NDArrayIndexer target_depth_indexer(target_depth, 2);

    NDArrayIndexer source_intensity_indexer(source_intensity, 2);
    NDArrayIndexer target_intensity_indexer(target_intensity, 2);

    NDArrayIndexer target_depth_dx_indexer(target_depth_dx, 2);
    NDArrayIndexer target_depth_dy_indexer(target_depth_dy, 2);
    NDArrayIndexer target_intensity_dx_indexer(target_intensity_dx, 2);
    NDArrayIndexer target_intensity_dy_indexer(target_intensity_dy, 2);

    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);

    core::Device device = source_vertex_map.GetDevice();
    core::Tensor trans = init_source_to_target;
    t::geometry::kernel::TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);
    const int64_t n = rows * cols;

    // A_29xN is a {29, N} shaped tensor, which is later reduced to
    // {29} where [0, 20] elements are used to construct {6,6} shaped
    // symmetric AtA matrix, [21, 26] elements are used to construct {6} AtB
    // matrix, element [27] stores residual and element [28] stores count.
    core::Tensor A_29xN =
            core::Tensor::Empty({29, n}, core::Dtype::Float32, device);
    float* A_reduction = A_29xN.GetDataPtr<float>();

    core::kernel::CUDALauncher::LaunchGeneralKernel(
            n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float J_I[6], J_D[6];
                float r_I, r_D;

                bool valid = GetJacobianHybrid(
                        workload_idx, cols, depth_diff, source_depth_indexer,
                        target_depth_indexer, source_intensity_indexer,
                        target_intensity_indexer, target_depth_dx_indexer,
                        target_depth_dy_indexer, target_intensity_dx_indexer,
                        target_intensity_dy_indexer, source_vertex_indexer, ti,
                        J_I, J_D, r_I, r_D);

                if (valid) {
                    for (int i = 0, j = 0; j < 6; j++) {
                        for (int k = 0; k <= j; k++) {
                            A_reduction[n * i + workload_idx] =
                                    J_I[j] * J_I[k] + J_D[j] * J_D[k];
                            i++;
                        }
                        A_reduction[n * (21 + j) + workload_idx] =
                                J_I[j] * r_I + J_D[j] * r_D;
                    }
                    A_reduction[n * 27 + workload_idx] = r_I * r_I + r_D * r_D;
                    A_reduction[n * 28 + workload_idx] = 1;
                } else {
                    for (int i = 0; i < 29; i++) {
                        A_reduction[n * i + workload_idx] = 0;
                    }
                }
            });
    ReduceAndSolve6x6(A_reduction, delta, residual, n, device);
}

}  // namespace odometry
}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
