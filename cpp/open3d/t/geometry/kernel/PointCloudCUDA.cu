#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include "open3d/t/geometry/kernel/PointCloudImpl.h"

namespace open3d {
namespace t {
namespace geometry {
namespace kernel {
namespace pointcloud {

void ProjectCUDA(
        core::Tensor& depth,
        utility::optional<std::reference_wrapper<core::Tensor>> image_colors,
        const core::Tensor& points,
        utility::optional<std::reference_wrapper<const core::Tensor>> colors,
        const core::Tensor& intrinsics,
        const core::Tensor& extrinsics,
        float depth_scale,
        float depth_max) {
    const bool has_colors = image_colors.has_value();

    int64_t n = points.GetLength();

    const float* points_ptr = points.GetDataPtr<float>();
    const float* point_colors_ptr =
            has_colors ? colors.value().get().GetDataPtr<float>() : nullptr;

    TransformIndexer transform_indexer(intrinsics, extrinsics, 1.0f);
    NDArrayIndexer depth_indexer(depth, 2);

    // Pass 1: depth map
    core::ParallelFor(
            depth.GetDevice(), n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float x = points_ptr[3 * workload_idx + 0];
                float y = points_ptr[3 * workload_idx + 1];
                float z = points_ptr[3 * workload_idx + 2];

                // coordinate in camera (in voxel -> in meter)
                float xc, yc, zc, u, v;
                transform_indexer.RigidTransform(x, y, z, &xc, &yc, &zc);

                // coordinate in image (in pixel)
                transform_indexer.Project(xc, yc, zc, &u, &v);
                if (!depth_indexer.InBoundary(u, v) || zc <= 0 ||
                    zc > depth_max) {
                    return;
                }

                float* depth_ptr = depth_indexer.GetDataPtr<float>(
                        static_cast<int64_t>(u), static_cast<int64_t>(v));
                float d = zc * depth_scale;
                float d_old = atomicExch(depth_ptr, d);
                if (d_old > 0) {
                    atomicMinf(depth_ptr, d_old);
                }
            });

    // Pass 2: color map
    if (!has_colors) return;

    NDArrayIndexer color_indexer(image_colors.value().get(), 2);
    float precision_bound = depth_scale * 1e-4;
    core::ParallelFor(
            depth.GetDevice(), n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float x = points_ptr[3 * workload_idx + 0];
                float y = points_ptr[3 * workload_idx + 1];
                float z = points_ptr[3 * workload_idx + 2];

                // coordinate in camera (in voxel -> in meter)
                float xc, yc, zc, u, v;
                transform_indexer.RigidTransform(x, y, z, &xc, &yc, &zc);

                // coordinate in image (in pixel)
                transform_indexer.Project(xc, yc, zc, &u, &v);
                if (!depth_indexer.InBoundary(u, v) || zc <= 0 ||
                    zc > depth_max) {
                    return;
                }

                float dmap = *depth_indexer.GetDataPtr<float>(
                        static_cast<int64_t>(u), static_cast<int64_t>(v));
                float d = zc * depth_scale;
                if (d < dmap + precision_bound) {
                    uint8_t* color_ptr = color_indexer.GetDataPtr<uint8_t>(
                            static_cast<int64_t>(u), static_cast<int64_t>(v));
                    color_ptr[0] = static_cast<uint8_t>(
                            point_colors_ptr[3 * workload_idx + 0] * 255.0);
                    color_ptr[1] = static_cast<uint8_t>(
                            point_colors_ptr[3 * workload_idx + 1] * 255.0);
                    color_ptr[2] = static_cast<uint8_t>(
                            point_colors_ptr[3 * workload_idx + 2] * 255.0);
                }
            });
}

template <typename scalar_t>
__global__ void EstimateNormalsFromCovariancesCUDAKernel(
        const scalar_t* covariances_ptr,
        scalar_t* normals_ptr,
        const bool has_normals,
        const int64_t n) {
    const int64_t workload_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (workload_idx >= n) return;

    int64_t covariances_offset = 9 * workload_idx;
    int64_t normals_offset = 3 * workload_idx;
    scalar_t normals_output[3] = {0};
    EstimatePointWiseNormalsWithFastEigen3x3<scalar_t>(
            covariances_ptr + covariances_offset, normals_output);

    if ((normals_output[0] * normals_output[0] +
         normals_output[1] * normals_output[1] +
         normals_output[2] * normals_output[2]) == 0.0 &&
        !has_normals) {
        normals_output[0] = 0.0;
        normals_output[1] = 0.0;
        normals_output[2] = 1.0;
    }
    if (has_normals) {
        if ((normals_ptr[normals_offset] * normals_output[0] +
             normals_ptr[normals_offset + 1] * normals_output[1] +
             normals_ptr[normals_offset + 2] * normals_output[2]) < 0.0) {
            normals_output[0] *= -1;
            normals_output[1] *= -1;
            normals_output[2] *= -1;
        }
    }

    normals_ptr[normals_offset] = normals_output[0];
    normals_ptr[normals_offset + 1] = normals_output[1];
    normals_ptr[normals_offset + 2] = normals_output[2];
}

void EstimateNormalsFromCovariancesCUDA(const core::Tensor& covariances,
                                        core::Tensor& normals,
                                        const bool has_normals) {
    core::Dtype dtype = covariances.GetDtype();
    int64_t n = covariances.GetLength();

    const dim3 blocks((n + 512 - 1) / 512);
    const dim3 threads(512);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        EstimateNormalsFromCovariancesCUDAKernel<<<blocks, threads, 0,
                                                   core::cuda::GetStream()>>>(
                covariances.GetDataPtr<scalar_t>(),
                normals.GetDataPtr<scalar_t>(), has_normals, n);
    });
}

}  // namespace pointcloud
}  // namespace kernel
}  // namespace geometry
}  // namespace t
}  // namespace open3d
